#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <cmath>


#include <coreutils/classes/matrixes/Matrix3D.cuh>

#include <coreutils/functions/debug/print.hpp>
#include <coreutils/functions/sort/sortHelpers.hpp>
#include <coreutils/functions/math/simpleMath.hpp>

#include <coreutils/util/time.hpp>
#include <coreutils/util/cudaErrors.cuh>

using namespace coreutils::functions;
using namespace coreutils::classes::matrixes;

int Matrix3D::getLength () {
	return this->length;
}

int Matrix3D::getWidth () {
	return this->width;
}

int Matrix3D::getHeight () {
	return this->height;
}

float* Matrix3D::getArr () {
	return arr;
}

long long Matrix3D::getSize () {
	return memorySize;
}

float* Matrix3D::getData (int length, int width, int height) {
	if (this->length <= length || this->width <= width || this->height <= height) {
		std::cout << "Invalid input at getData";
		return nullptr;
	}
	return &this->arr[getIndex(length, width, height)];
}

int Matrix3D::getIndex (int l, int w, int h) const {
	return l * this->width * this->height + w * this->height + h;
}

void Matrix3D::shuffleEvery () {
	srand(GetTimeStamp().tv_sec + GetTimeStamp().tv_usec);
	for (int length = 0; length < this->length; length++) {
		for (int width = 0; width < this->width; width++) {
			for (int height = 0; height < this->height; height++) {
				sort::swap (&this->arr[getIndex(length, width, height)], 
								&this->arr[getIndex((double) rand() / RAND_MAX * length, (double) rand() / RAND_MAX * width, (double) rand() / RAND_MAX * height)]);

			}
		}
	}
}


int* Matrix3D::shuffleGroups () {
	int* order = new int[this->length];
	for (int length = 0; length < this->length; length++) {
		srand(GetTimeStamp().tv_sec + GetTimeStamp().tv_usec);
		double randomLength = rand() / RAND_MAX * length;
		order[length] = randomLength;
		for (int width = 0; width < this->width; width++) {
			for (int height = 0; height < this->height; height++) {
				sort::swap (&this->arr [getIndex(length, width, height)], 
								&this->arr [getIndex(randomLength, width, height)]);

			}
		}
	}
	return order;
}

void Matrix3D::shuffleGroups (int* order) {
	for (int length = 0; length < this->length; length++) {
		for (int width = 0; width < this->width; width++) {
			for (int height = 0; height < this->height; height++) {
				sort::swap (&this->arr [getIndex(length, width, height)], 
								&this->arr [getIndex(order[length], width, height)]);

			}
		}
	}
}

void Matrix3D::operator += (const Matrix3D* m2) {
	for (int i = 0; i < length; i++) {
		for (int j = 0; j < width; j++) {
			for (int k = 0; k < height; k++) {
				this->arr [getIndex(i, j, k)] += m2->arr [getIndex(i, j, k)];
			}
		}
	}
}

void Matrix3D::operator -= (const Matrix3D* m2) {
	for (int i = 0; i < length; i++) {
		for (int j = 0; j < width; j++) {
			for (int k = 0; k < height; k++) {
				this->arr [getIndex(i, j, k)] -= m2->arr [getIndex(i, j, k)];
			}
		}
	}
}

Matrix3D* Matrix3D::operator + (const Matrix3D* m2) {
	Matrix3D* M3D = new Matrix3D (this->length, this->width, this->height);

	for (int i = 0; i < length; i++) {
		for (int j = 0; j < width; j++) {
			for (int k = 0; k < height; k++) {
				M3D->arr[getIndex(i, j, k)] = this->arr [getIndex(i, j, k)] + m2->arr [getIndex(i, j, k)];
			}
		}
	}

	return M3D;
}

Matrix3D* Matrix3D::operator - (const Matrix3D* m2) {
	Matrix3D* M3D = new Matrix3D (this->length, this->width, this->height);

	for (int i = 0; i < length; i++) {
		for (int j = 0; j < width; j++) {
			for (int k = 0; k < height; k++) {
				M3D->arr[getIndex(i, j, k)] = this->arr [getIndex(i, j, k)] - m2->arr [getIndex(i, j, k)];
			}
		}
	}

	return M3D;
}

Matrix3D* Matrix3D::operator * (const Matrix3D* m2) {
	Matrix3D* M3D = new Matrix3D(this->length, this->width, this->height);

	for (int i = 0; i < length; i++) {
		for (int j = 0; j < width; j++) {
			for (int k = 0; k < height; k++) {
				M3D->arr[getIndex(i, j, k)] = this->arr [getIndex(i, j, k)] * m2->arr [getIndex(i, j, k)];
			}
		}
	}

	return M3D;
}

Matrix3D* Matrix3D::operator * (const float x) {
	Matrix3D* M3D = new Matrix3D (this->length, this->width, this->height);

	for (int i = 0; i < length; i++) {
		for (int j = 0; j < width; j++) {
			for (int k = 0; k < height; k++) {
				M3D->arr[getIndex(i, j, k)] = this->arr [getIndex(i, j, k)] * x;
			}
		}
	}
	
	return M3D;
}

Matrix3D* Matrix3D::operator / (const Matrix3D* m2) {
	Matrix3D* M3D = new Matrix3D (this->length, this->width, this->height);

	for (int i = 0; i < length; i++) {
		for (int j = 0; j < width; j++) {
			for (int k = 0; k < height; k++) {
				M3D->arr[getIndex(i,j,k)] = this->arr [getIndex(i,j,k)] / m2->arr [getIndex(i,j,k)];
			}
		}
	}
	
	return M3D;
}

bool Matrix3D::equals (const Matrix3D* m2, double tolerance) {
	for (int i = 0; i < length; i++) {
		for (int j = 0; j < width; j++) {
			for (int k = 0; k < height; k++) {
				if (m2->arr[getIndex(i, j, k)] - this->arr [getIndex(i, j, k)] > tolerance) {
					return false;
				}
			}
		}
	}

	return true;
}

void Matrix3D::randomize (double lowerBound, double upperBound) {
	double currentRandomNumber;
	srand(GetTimeStamp().tv_sec + GetTimeStamp().tv_usec);
	for (int i = 0; i < this->length; i++) {
		for (int j = 0; j < this->width; j++) {
			for (int k = 0; k < this->height; k++) {
				currentRandomNumber = ((double) rand()) / RAND_MAX * (upperBound - lowerBound) + lowerBound;
				this->arr [getIndex(i, j, k)] = currentRandomNumber;
			}
		}
	}
}

void Matrix3D::xavierRandomize (int l1, int w1, int h1, int l2, int w2, int h2) {
	double bound = sqrt(6) / (sqrt(l1 * w1 * h1 + l2 * w2 * h2));
	this->randomize(-bound, bound);
}

double Matrix3D::dotProduct (const Matrix3D* m2) {
	double output = 0;
	
	for (int i = 0; i < length; i++) {
		for (int j = 0; j < width; j++) {
			for (int k = 0; k < height; k++) {
				output += this->arr [getIndex(i, j, k)] * m2->arr [getIndex(i, j, k)];
			}
		}
	}
	
	return output;
}

double Matrix3D::sum () {
	double output = 0;
	
	for (int i = 0; i < length; i++) {
		for (int j = 0; j < width; j++) {
			for (int k = 0; k < height; k++) {
				output += this->arr [getIndex(i, j, k)];
			}
		}
	}
	
	return output;
}

void Matrix3D::insert (float data, int length, int width, int height) {
	this->arr[getIndex(length, width, height)] = data;
}

void Matrix3D::printMatrix () const {
	std::cout << '\n' << "{";
	for (int i = 0; i < this->length; i++) {
		std::cout << '\n' << "  {" << '\n';
		for (int j = 0; j < this->width; j++) {
			std::string out = "    {";
			for (int k = 0; k < this->height; k++) {
				out += std::to_string(this->arr [getIndex(i, j, k)]) + ", ";
			}
			out = out.substr(0, out.length () - 2);
			std::cout << out << "}" << '\n';
		}
		std::cout << "  }";
	}
	std::cout << '\n' << "}" << '\n';
}

void Matrix3D::setMatrix (Matrix3D* M3D) {
	for (int i = 0; i < length; i++) {
		for (int j = 0; j < width; j++) {
			for (int k = 0; k < height; k++) {
				this->insert(*M3D->getData(i, j, k), i, j, k);
			}
		}
	}
}

void Matrix3D::setAll (double x) {
	for (int i = 0; i < length; i++) {
		for (int j = 0; j < width; j++) {
			for (int k = 0; k < height; k++) {
				this->insert(x, i, j, k);
			}
		}
	}
}

Matrix3D::Matrix3D (const int length, const int width, const int height) {
	this->length = length;
	this->width = width;
	this->height = height;
	gpuErrchk(hipHostMalloc((void **) &this->arr, length * width * height * sizeof(float)));
	this->memorySize = length * width * height * sizeof(float);
}

Matrix3D::Matrix3D (const Matrix3D &m3d) {
	this->length = m3d.length;
	this->width = m3d.width;
	this->height = m3d.height;
	gpuErrchk(hipHostMalloc((void **) &this->arr, length * width * height * sizeof(float)));
	gpuErrchk(hipMemcpy(this->arr, m3d.arr, this->length * this->width * this->height * sizeof(float), hipMemcpyHostToHost));
	this->memorySize = length * width * height * sizeof(float);
}

Matrix3D::Matrix3D () {
	this->length = 0;
	this->width = 0;
	this->height = 0;
	this->arr = nullptr;
	this->memorySize = 0;
}

Matrix3D::~Matrix3D () {
	hipHostFree(this->arr);
}